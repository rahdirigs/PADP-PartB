#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <opencv2/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <string>
#include <utility>

const int BLOCK_SIZE = 16;
const int FILTER_HEIGHT = 3;
const int FILTER_WIDTH = 3;

using namespace std;
using namespace cv;

pair<string, string> process_file_name(const string &input_image) {
  int len = (int) input_image.length();
  int dot = -1;
  for (int i = 0; i < len; ++i) {
    if (input_image[i] == '.') {
      dot = i;
      break;
    }
  }
  if (dot == -1) {
    cerr << "Invalid file name\n";
    exit(0);
  }
  string name = input_image.substr(0, dot);
  string ext = input_image.substr(dot + 1, len - dot - 1);
  return make_pair(name, ext);
}

__device__ void bubble_sort(unsigned char *values) {
  int n = FILTER_WIDTH * FILTER_HEIGHT;
  for (int i = 0; i < n; ++i) {
    for (int j = i + 1; j < n; ++j) {
      if (values[i] > values[j]) {
        unsigned char temp = values[i];
        values[i] = values[j];
        values[j] = temp;
      }
    }
  }
}

__global__ void median_filter(unsigned char *img, unsigned char *omg, unsigned int width, unsigned int height, int channels) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  bool filter_inside_image = (x >= FILTER_WIDTH / 2) && (x < width - FILTER_WIDTH / 2) && (y >= FILTER_HEIGHT / 2) && (y < height - FILTER_HEIGHT / 2);
  if (filter_inside_image) {
    for (int channel = 0; channel < channels; ++channel) {
      unsigned char values[FILTER_HEIGHT * FILTER_WIDTH];
      for (int i = -FILTER_HEIGHT / 2; i <= FILTER_HEIGHT / 2; ++i) {
        for (int j = -FILTER_WIDTH / 2; j <= FILTER_WIDTH / 2; ++j) {
          values[i * FILTER_WIDTH + j] = img[((y + i) * width + (x + j)) * channels + channel];
        }
      }
      bubble_sort(values);
      int median = FILTER_WIDTH * FILTER_HEIGHT / 2;
      omg[(y * width + x) * channels + channel] = values[median];
    }
  }
}

void filter(Mat &img, Mat &omg) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int channels = img.step / img.cols;
  const int input_size = img.rows * img.cols * channels;
  const int output_size = omg.rows * omg.cols * channels;
  unsigned char *d_img, *d_omg;
  hipMalloc<unsigned char>(&d_img, input_size);
  hipMalloc<unsigned char>(&d_omg, output_size);
  hipMemcpy(d_img, img.ptr(), input_size, hipMemcpyHostToDevice);
  const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  const dim3 grid((omg.cols + block.x - 1) / block.x, (omg.rows + block.y - 1) / block.y);
  hipEventRecord(start);
  median_filter<<<grid, block>>>(d_img, d_omg, omg.cols, omg.rows, channels);
  hipEventRecord(stop);
  hipMemcpy(omg.ptr(), d_omg, output_size, hipMemcpyDeviceToHost);
  hipFree(d_img);
  hipFree(d_omg);
  hipEventSynchronize(stop);
  float time_elapsed = 0.0;
  hipEventElapsedTime(&time_elapsed, start, stop);
  cout << "Time taken for execution = " << time_elapsed << "ms.\n";
}

int main(int argc, char **argv) {
  if (argc != 2) {
    cerr << "Usage: ./main <input_image>";
    return 0;
  }
  string input_image = string(argv[1]);
  pair<string, string> name_extension = process_file_name(input_image);
  string input_image_name = name_extension.first;
  string input_image_ext = name_extension.second;
  string output_image = input_image_name + "_out." + input_image_ext;
  Mat img = imread(input_image, CV_LOAD_IMAGE_UNCHANGED);
  if (img.empty()) {
    cerr << "No image found " << input_image << "\n";
    return 0;
  }
  cout << "Input image dimensions:\n";
  cout << "Rows = " << img.rows << "\nColumns = " << img.cols << "\nChannels = " << img.channels() << "\n";
  Mat omg(img.size(), img.type());
  filter(img, omg);
  imwrite(output_image, omg);
  return 0;
}
