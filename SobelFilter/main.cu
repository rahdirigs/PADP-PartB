#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <opencv2/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <string>
#include <utility>

const int BLOCK_SIZE = 16;
const int FILTER_HEIGHT = 3;
const int FILTER_WIDTH = 3;

using namespace std;
using namespace cv;

pair<string, string> process_file_name(const string &input_image) {
  int len = (int) input_image.length();
  int dot = -1;
  for (int i = 0; i < len; ++i) {
    if (input_image[i] == '.') {
      dot = i;
      break;
    }
  }
  if (dot == -1) {
    cerr << "Invalid file name\n";
    exit(0);
  }
  string name = input_image.substr(0, dot);
  string ext = input_image.substr(dot + 1, len - dot - 1);
  return make_pair(name, ext);
}

__global__ void sobel_filter(unsigned char *img, unsigned char *omg, unsigned int width, unsigned int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  const float Gx[FILTER_HEIGHT][FILTER_WIDTH] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1}
  };
  const float Gy[FILTER_HEIGHT][FILTER_WIDTH] = {
    {1, 2, 1},
    {0, 0, 0},
    {-1, -2, -1}
  };
  bool filter_inside_image = (x >= FILTER_WIDTH / 2) && (x < width - FILTER_WIDTH / 2) && (y >= FILTER_HEIGHT / 2) && (y < height - FILTER_HEIGHT / 2);
  if (filter_inside_image) {
    float gx = 0.0;
    for (int i = -FILTER_HEIGHT / 2; i <= FILTER_HEIGHT / 2; ++i) {
      for (int j = -FILTER_WIDTH / 2; j <= FILTER_WIDTH / 2; ++j) {
        float value = img[(y + i) * width + (x + j)];
        gx += value * Gx[i + FILTER_HEIGHT / 2][j + FILTER_WIDTH / 2];
      }
    }
    if (gx < 0.0) {
      gx *= -1.0;
    }
    omg[y * width + x] = gx;
    float gy = 0.0;
    for (int i = -FILTER_HEIGHT / 2; i <= FILTER_HEIGHT / 2; ++i) {
      for (int j = -FILTER_WIDTH / 2; j <= FILTER_WIDTH / 2; ++j) {
        float value = img[(y + i) * width + (x + j)];
        gy += value * Gy[i + FILTER_HEIGHT / 2][j + FILTER_WIDTH / 2];
      }
    }
    if (gy < 0.0) {
      gy *= -1.0;
    }
    omg[y * width + x] = gy;
  }
}

void filter(Mat &img, Mat &omg) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  const int input_size = img.rows * img.cols;
  const int output_size = omg.rows * omg.cols;
  unsigned char *d_img, *d_omg;
  hipMalloc<unsigned char>(&d_img, input_size);
  hipMalloc<unsigned char>(&d_omg, output_size);
  hipMemcpy(d_img, img.ptr(), input_size, hipMemcpyHostToDevice);
  const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  const dim3 grid((omg.cols + block.x - 1) / block.x, (omg.rows + block.y - 1) / block.y);
  hipEventRecord(start);
  sobel_filter<<<grid, block>>>(d_img, d_omg, omg.cols, omg.rows);
  hipEventRecord(stop);
  hipMemcpy(omg.ptr(), d_omg, output_size, hipMemcpyDeviceToHost);
  hipFree(d_img);
  hipFree(d_omg);
  hipEventSynchronize(stop);
  float time_elapsed = 0.0;
  hipEventElapsedTime(&time_elapsed, start, stop);
  cout << "Time taken for execution = " << time_elapsed << "ms.\n";
}

int main(int argc, char **argv) {
  if (argc != 2) {
    cerr << "Usage: ./main <input_image>";
    return 0;
  }
  string input_image = string(argv[1]);
  pair<string, string> name_extension = process_file_name(input_image);
  string input_image_name = name_extension.first;
  string input_image_ext = name_extension.second;
  string output_image = input_image_name + "_out." + input_image_ext;
  Mat img = imread(input_image, CV_LOAD_IMAGE_UNCHANGED);
  if (img.empty()) {
    cerr << "No image found " << input_image << "\n";
    return 0;
  }
  cout << "Input image dimensions:\n";
  cout << "Rows = " << img.rows << "\nColumns = " << img.cols << "\n";
  cvtColor(img, img, CV_BGR2GRAY);
  Mat omg(img.size(), img.type());
  filter(img, omg);
  omg.convertTo(omg, CV_32F, 1.0 / 255, 0);
  omg *= 255;
  imwrite(output_image, omg);
  return 0;
}
